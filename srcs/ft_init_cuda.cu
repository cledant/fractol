#include "fract_ol.h"

static void		ft_init_extent(t_mlx *e)
{
	(e->fl_extent).width = sizeof(float) * e->win_x_size;
	(e->fl_extent).height = sizeof(float) * e->win_y_size;
	(e->fl_extent).depth = sizeof(float) * 5;
	(e->uint_extent).width = sizeof(unsigned int) * e->win_x_size;
	(e->uint_extent).height = sizeof(unsigned int) * e->win_y_size;
	(e->uint_extent).depth = sizeof(unsigned int) * 2;
}

static void		ft_clean(t_mlx *e, int error)
{
	if (error < 2)
		hipFree(e->fl_matrix);
	if (error < 3)
		hipFree(e->uint_matrix);
	if (error < 4)
		hipFree(e->d_x_min);
	if (error < 5)
		hipFree(e->d_y_max);
	if (error < 6)
		hipFree(e->d_x_pitch);
	if (error < 7)
		hipFree(e->d_y_pitch);
	if (error < 8)
		hipFree(e->d_y_pitch);
	if (error < 9)
		hipFree(e->d_iter);
}

int				ft_cuda_init(t_mlx *e)
{
	e->nb_block = (e->win_x_size % M_THREAD == 0) ? e->win_x_size % M_THREAD :
		e->win_x_size % M_THREAD + 1;
	ft_init_extent(e);
	if (hipMalloc3D(e->fl_matrix, e->fl_extent) != hipSuccess)
		return (0);
	if (hipMalloc3D(e->uint_matrix, e->uint_extent) != hipSuccess)
	{
		ft_clean(e, 1);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_x_min), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 2);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_y_max), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 3);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_x_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 4);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_y_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 5);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_buff_img), sizeof(int) * e->win_x_size *
					e->win_y_size) != hipSuccess)
	{
		ft_clean(e, 6);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_color), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 7);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_iter), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 8);
		return (0);
	}
	hipMemcpy(e->d_iter, (const void *)e->iter, sizeof(size_t),
			hipMemcpyHostToDevice);
	return (1);
}
