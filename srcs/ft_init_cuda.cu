# **************************************************************************** #
#                                                                              #
#                                                         :::      ::::::::    #
#    ft_init_cuda.cu                                    :+:      :+:    :+:    #
#                                                     +:+ +:+         +:+      #
#    By: cledant <cledant@student.42.fr>            +#+  +:+       +#+         #
#                                                 +#+#+#+#+#+   +#+            #
#    Created: 2016/07/26 21:55:33 by cledant           #+#    #+#              #
#    Updated: 2016/07/28 15:27:22 by cledant          ###   ########.fr        #
#                                                                              #
# **************************************************************************** #

#include "fract_ol.h"

static void		ft_init_extent(t_mlx *e)
{
	(e->fl_extent).width = sizeof(float) * e->win_x_size;
	(e->fl_extent).height = sizeof(float) * e->win_y_size;
	(e->fl_extent).depth = sizeof(float) * 5;
	(e->uint_extent).width = sizeof(unsigned int) * e->win_x_size;
	(e->uint_extent).height = sizeof(unsigned int) * e->win_y_size;
	(e->uint_extent).depth = sizeof(unsigned int) * 2;
}

static void		ft_clean(t_mlx *e, int error)
{
	if (error < 2)
		hipFree(e->fl_matrix);
	if (error < 3)
		hipFree(e->unit_matrix);
	if (error < 4)
		hipFree(e->d_x_min);
	if (error < 5)
		hipFree(e->d_y_max);
	if (error < 6)
		hipFree(e->d_x_pitch);
	if (error < 7)
		hipFree(e->d_y_pitch);
}

int				ft_cuda_init(t_mlx *e)
{
	ft_init_extent(e);
	if (hipMalloc3D(e->fl_matrix, e->fl_extent) != hipSuccess)
		return (0);
	if (hipMalloc3D(e->uint_matrix, e->uint_extent) != hipSuccess)
	{
		ft_clean(1);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_x_min), sizeof(float)) != hipSuccess)
	{
		ft_clean(2);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_y_max), sizeof(float)) != hipSuccess)
	{
		ft_clean(3);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_x_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(4);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_y_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(5);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_buff_img), sizeof(int) * e->win_x_size *
					e->win_y_size) != hipSuccess)
	{
		ft_clean(6);
		return (0);
	}
	return (1);
}
