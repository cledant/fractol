#include "fract_ol.h"

static void		ft_clean(t_mlx *e, int error)
{
	if (error < 2)
		hipFree(e->d_y_max);
	if (error < 3)
		hipFree(e->d_x_pitch);
	if (error < 4)
		hipFree(e->d_y_pitch);
	if (error < 5)
		hipFree(e->d_buff_img);
	if (error < 6)
		hipFree(e->d_color);
	if (error < 7)
		hipFree(e->d_iter);
	if (error < 8)
		hipFree(e->d_win_x_size);
	if (error < 9)
		hipFree(e->d_win_y_size);
}

int				ft_cuda_init(t_mlx *e)
{
	e->nb_block = (e->win_x_size % M_THREAD == 0) ? e->win_x_size % M_THREAD :
		e->win_x_size % M_THREAD + 1;
	if (hipMalloc((void **)(&e->d_x_min), sizeof(float)) != hipSuccess)
		return (0);
	if (hipMalloc((void **)(&e->d_y_max), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 1);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_x_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 2);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_y_pitch), sizeof(float)) != hipSuccess)
	{
		ft_clean(e, 3);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_buff_img), sizeof(int) * e->win_x_size *
					e->win_y_size) != hipSuccess)
	{
		ft_clean(e, 4);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_color), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 5);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_iter), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 6);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_win_x_size), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 7);
		return (0);
	}
	if (hipMalloc((void **)(&e->d_win_y_size), sizeof(size_t)) != hipSuccess)
	{
		ft_clean(e, 8);
		return (0);
	}
	hipMemcpy(e->d_iter, (const void *)e->iter, sizeof(size_t),
			hipMemcpyHostToDevice);
	return (1);
}
