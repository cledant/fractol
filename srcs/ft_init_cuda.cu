# **************************************************************************** #
#                                                                              #
#                                                         :::      ::::::::    #
#    ft_init_cuda.cu                                    :+:      :+:    :+:    #
#                                                     +:+ +:+         +:+      #
#    By: cledant <cledant@student.42.fr>            +#+  +:+       +#+         #
#                                                 +#+#+#+#+#+   +#+            #
#    Created: 2016/07/26 21:55:33 by cledant           #+#    #+#              #
#    Updated: 2016/07/26 22:31:18 by cledant          ###   ########.fr        #
#                                                                              #
# **************************************************************************** #

#include "fract_ol.h"

static void		ft_init_fb_ext(t_mlx *e)
{
	(e->fl_extent).width = sizeof(float) * e->win_x_size;
	(e->fl_extent).height = sizeof(float) * e->win_y_size;
	(e->fl_extent).depth = sizeof(float) * 5;
	(e->uint_extent).width = sizeof(unsigned int) * e->win_x_size;
	(e->uint_extent).height = sizeof(unsigned int) * e->win_y_size;
	(e->uint_extent).depth = sizeof(unsigned int) * 2;
}

int				ft_cuda_init(t_mlx *e)
{
	ft_init_ext(e);
	if (hipMalloc3D(e->fl_matrix, e->fl_extent) != hipSuccess)
		return (0);
	if (hipMalloc3D(e->uint_matrix, e->uint_extent) != hipSuccess)
	{
		hipFree(e->fl_matrix);
		return (0);
	}
	hipMalloc((void **)(&e->d_x_min), sizeof(float));
	return (1);
}
