/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_mlx_i_draw_mandelbrot.c                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: cledant <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/01/19 12:35:05 by cledant           #+#    #+#             */
/*   Updated: 2016/01/22 18:48:01 by cledant          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" 
{
#include "fract_ol.h"
}

extern "C"
void		ft_mlx_i_draw_cuda(t_mlx *e)
{
	dim3 thread2d(M_THREAD_X , M_THREAD_Y);
	dim3 block2d(e->nb_block_x, e->nb_block_y);

	hipMemcpy(e->d_iter, (const void *)&e->iter, sizeof(size_t),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_win_x_size, (const void *)&e->win_x_size, sizeof(size_t),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_win_y_size, (const void *)&e->win_y_size, sizeof(size_t),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_min, (const void *)&e->x_min, sizeof(double),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_max, (const void *)&e->y_max, sizeof(double),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_pitch, (const void *)&e->x_pitch, sizeof(double),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_pitch, (const void *)&e->y_pitch, sizeof(double),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_color, (const void *)&e->color, sizeof(size_t),
			hipMemcpyHostToDevice);
	if (e->fractal == 1)
	{
		ft_matrix_calc_mb<<<block2d, thread2d>>>(e->d_buff_img,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_win_x_size,
			e->d_win_y_size, e->d_color, e->d_iter);
	}
	else if (e->fractal == 3)
	{
		ft_matrix_calc_bs<<<block2d, thread2d>>>(e->d_buff_img,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_win_x_size,
			e->d_win_y_size, e->d_color, e->d_iter);
	}
	hipMemcpy(e->buff_img, (const void *)e->d_buff_img,
			e->win_x_size * e->win_y_size * sizeof(int), hipMemcpyDeviceToHost);
}
