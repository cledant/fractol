#include "hip/hip_runtime.h"
#include "fract_ol.h"

__global__ void		ft_copy_to_image(struct hipPitchedPtr *uint_matrix,
						char *buff, size_t *win_x)
{
	size_t	idx_x;
	size_t	idx_y;

	cpy_buff = buff;
	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	cpy_buff = cpy_buff + idx_x * 4 + idx_y * win_x;
	hipMemcpy(buff, uint_matrix[idx_x][idx_y][1], sizeof(int),
			hipMemcpyDeviceToDevice);
}
