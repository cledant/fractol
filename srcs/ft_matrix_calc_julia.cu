#include "hip/hip_runtime.h"
extern "C"
{
#include "fract_ol.h"
}

__device__ static size_t		ft_julia_it(double *pos_real, double init[2],
									size_t *it_max)
{
	size_t	it;
	double	sqrt[2];
	double	tmp[2];

	it = 0;
	while (1)
	{
		sqrt[0] = init[0] * init[0];
		sqrt[1] = init[1] * init[1];
		tmp[0]= sqrt[0] - sqrt[1] + pos_real[0];
		tmp[1] = init[0] * init[1];
		tmp[1] += tmp[1];
		tmp[1] += pos_real[1];
		init[0] = tmp[0]; 
		init[1] = tmp[1];
		if ((init[0] * init[0]) + (init[1] * init[1]) <= 4)
			it++;
		else
			return (it);
		if (it > *it_max)
			return (it);
	}
}

__global__ void			ft_matrix_calc_julia(unsigned int *color_buff,
							double *x_min, double *y_max, double *x_pitch,
							double *y_pitch, size_t *win_x_size,
							size_t *win_y_size, size_t *color, size_t *it_max,
							double *mouse)
{
	size_t	idx_x;
	size_t	idx_y;
	double	pos_real[2];
	double	init[2];
	size_t	it;

	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x < *win_x_size && idx_y < *win_y_size)
	{
		init[0] = *x_min + (idx_x * *x_pitch);
		init[1] = *y_max - (idx_y * *y_pitch);
		it = ft_julia_it(mouse, init, it_max);
		color_buff[idx_x + idx_y * *win_x_size] = ft_calc_color(it, color,
						it_max);
	}
}
