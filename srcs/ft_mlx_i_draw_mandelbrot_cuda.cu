/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_mlx_i_draw_mandelbrot.c                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: cledant <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/01/19 12:35:05 by cledant           #+#    #+#             */
/*   Updated: 2016/01/22 18:48:01 by cledant          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" 
{
#include "fract_ol.h"
}

extern "C"
void		ft_mlx_i_draw_mandelbrot_cuda(t_mlx *e)
{
	hipMemcpy(e->d_x_min, (const void *)&e->x_min, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_max, (const void *)&e->y_max, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_pitch, (const void *)&e->x_pitch, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_pitch, (const void *)&e->y_pitch, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_color, (const void *)&e->color, sizeof(size_t),
			hipMemcpyHostToDevice);
	ft_matrix_calc_mb<<<e->nb_block, M_THREAD>>>(e->d_buff_img,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_win_x_size,
			e->d_win_y_size, e->d_color, e->d_iter);
	hipMemcpy(e->buff_img, (const void *)e->d_buff_img,
			e->win_x_size * e->win_y_size * sizeof(int), hipMemcpyDeviceToHost);
}
