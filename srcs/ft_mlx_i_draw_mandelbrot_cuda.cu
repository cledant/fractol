/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_mlx_i_draw_mandelbrot.c                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: cledant <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/01/19 12:35:05 by cledant           #+#    #+#             */
/*   Updated: 2016/01/22 18:48:01 by cledant          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "fract_ol.h"

void		ft_mlx_i_draw_mandelbrot_cuda(t_mlx *e)
{
	hipMemcpy(e->d_x_min, (const void *)&e->x_min, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_max, (const void *)&e->y_max, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_pitch, (const void *)&e->x_pitch, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_pitch, (const void *)&e->y_pitch, sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(e->d_color, (const void *)e->color, sizeof(size_t),
			hipMemcpyHostToDevice);
	ft_matrix_calc_mb<<<e->nb_block, M_THREAD>>>(e->fl_matrix, e->uint_matrix,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_win_x_size,
			e->d_win_y_size, e->d_color, e->d_iter);
	ft_copy_to_image<<<e->nb_block, M_THREAD>>>(e->uint_matrix, e->d_buff_img,
			e->d_win_x_size);
	hipMemcpy(e->buff_img, (const void *)e->d_buff_img,
			e->win_x_size * e->win_y_size * 4, hipMemcpyDeviceToHost);
}
