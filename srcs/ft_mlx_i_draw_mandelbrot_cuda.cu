/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_mlx_i_draw_mandelbrot.c                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: cledant <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/01/19 12:35:05 by cledant           #+#    #+#             */
/*   Updated: 2016/01/22 18:48:01 by cledant          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "fract_ol.h"

void		ft_mlx_i_draw_mandelbrot_cuda(t_mlx *e)
{
	hipMemcpy(e->d_x_min, e->x_min, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_max, e->y_max, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_pitch, e->x_pitch, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_pitch, e->y_pitch, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_color, e->color, sizeof(size_t), hipMemcpyHostToDevice);
	ft_matrix_calc_mb<<<e->nb_block, M_THREAD>>>(e->ft_matrix, e->uint_matrix,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_color,
			e->d_iter);
	ft_copy_to_image<<<e->nb_block, M_THREAD>>>(e->uint_matrix, e->d_buff_img,
			e->d_win_x_size);
	hipMemcpy(e->buff_img, e->d_buff_img, e->win_x_size * e->win_y_size * 4,
			hipMemcpyDeviceToHost);
}
