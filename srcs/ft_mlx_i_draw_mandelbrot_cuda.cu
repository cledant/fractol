/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ft_mlx_i_draw_mandelbrot.c                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: cledant <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/01/19 12:35:05 by cledant           #+#    #+#             */
/*   Updated: 2016/01/22 18:48:01 by cledant          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

#include "fract_ol.h"

void		ft_mlx_i_draw_mandelbrot_cuda(t_mlx *e)
{
	hipMemcpy(e->d_x_min, e->x_min, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_max, e->y_max, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_x_pitch, e->x_pitch, sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(e->d_y_pitch, e->y_pitch, sizeof(float), hipMemcpyHostToDevice);
	ft_matrix_seek_pixel_color<<<cc, cc>>>(e->ft_matrix, e->uint_matrix,
			e->d_x_min, e->d_y_max, e->d_x_pitch, e->d_y_pitch, e->d_color);
	ft_copy_to_image<<<sdfswfd,sdfsdf>>>(e->uint_matrix, e->d_buff_img);
	hipMemcpy(e->buff_img, e->d_buff_img, e->win_x_size * e->win_y_size * 4,
			hipMemcpyDeviceToHost);
}
