#include "hip/hip_runtime.h"
extern "C"
{
#include "fract_ol.h"
#include <stdio.h>
}

__device__ static size_t		ft_mb_it(double pos_real[2], size_t *it_max)
{
	size_t	it;
	double	sqrt[2];
	double	tmp[2];
	double	init[2];

	it = 0;
	init[0] = 0;
	init[1] = 0;
	while (1)
	{
		sqrt[0] = init[0] * init[0];
		sqrt[1] = init[1] * init[1];
		tmp[0]= sqrt[0] - sqrt[1] + pos_real[0];
		tmp[1] = init[0] * init[1];
		tmp[1] += tmp[1];
		tmp[1] += pos_real[1];
		init[0] = tmp[0]; 
		init[1] = tmp[1];
		if ((init[0] * init[0]) + (init[1] * init[1]) <= 4)
			it++;
		else
			return (it);
		if (it > *it_max)
			return (it);
	}
}

__device__ static unsigned int	ft_calc_color(size_t it, size_t *color, 
									size_t *it_max)
{
	if (it <= *it_max)
	{
		if (*color == 1)
			return (it * 0x000FF000 + 0x000000FF);
		else if (*color == 2)
			return (it * 0x00102030 + 0x000F0F0F);
		else if (*color == 3)
			return (it * 0x00111111 + 0x00F0000F);
		else if (*color == 4)
			return (it * 0x00060006 + 0x00000000);
		else if (*color == 5)
			return (it * 0x00341165 + 0x00A4B580);
		else if (*color == 6)
			return (it * 0x00060100 + 0x00000000);
	}
	else
		return (0x00000000);
	return (0x00000000);
}

__global__ void			ft_matrix_calc_mb(unsigned int *color_buff,
							double *x_min, double *y_max, double *x_pitch,
							double *y_pitch, size_t *win_x_size,
							size_t *win_y_size, size_t *color, size_t *it_max)
{
	size_t	idx_x;
	size_t	idx_y;
	double	pos_real[2];
	size_t	it;

	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x < *win_x_size && idx_y < *win_y_size)
	{
		pos_real[0] = *x_min + (idx_x * *x_pitch);
		pos_real[1] = *y_max - (idx_y * *y_pitch);
		it = ft_mb_it(pos_real, it_max);
		color_buff[idx_x + idx_y * *win_x_size] = ft_calc_color(it, color,
						it_max);
	}
}
