#include "hip/hip_runtime.h"
#include "fract_ol.h"

__device__ static size_t		ft_mb_it(float pos_real[2], size_t *it_max)
{
	size_t	it;
	float	sqrt[2];
	float	tmp[2];
	float	init[2];

	it = 0;
	init[0] = 0;
	init[1] = 0;
	while (1)
	{
		sqrt[0] = init[0] * init[0];
		sqrt[1] = init[1] * init[1];
		tmp[0]= sqrt[0] - sqrt[1] + pos_real[0];
		tmp[1] = sqrt[0] * sqrt[1];
		tmp[1] += tmp[1];
		tmp[1] += pos_real[1];
		init[0] = tmp[0]; 
		init[1] = tmp[1];
		if ((init[0] * init[0]) + (init[1] * init[1]) <= 4)
			return (it);
		else
			it++;
		if (it > *it_max)
			return (it);
	}
}

__device__ static unsigned int	ft_calc_color(size_t it, int *color, 
									size_t *it_max)
{
	if (it <= *it_max)
	{
		if (*color == 1)
			return (it * 0x000FF000 + 0x000000FF);
		else if (*color == 2)
			return (it * 0x000F0F0F + 0x00102030);
		else if (*color == 3)
			return (it * 0x00F0000F + 0x00111111);
		else if (*color == 4)
			return (it * 0x00000000 + 0x00060006);
		else if (*color == 5)
			return (it * 0x00A4B580 + 0x00341165);
		else if (*color == 6)
			return (it * 0x00000000 + 0x00060100);
	}
	else
		return (0x00000000);
	return (0x00000000);
}

__global__ void			ft_matrix_calc_mb(unsigned int *color_buff,
							float *x_min, float *y_max, float *x_pitch,
							float *y_pitch, size_t *win_x_size,
							size_t *win_y_size, int *color, size_t *it_max)
{
	size_t	idx_x;
	size_t	idx_y;
	float	pos_real[2];
	size_t	it;

	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x < *win_x_size && idx_y < *win_y_size)
	{
		pos_real[0] = *x_min + (idx_x * *x_pitch);
		pos_real[1] = *y_max + (idx_y * *y_pitch);
		it = ft_mb_it(pos_real, *it_max);
		color_buff[idx_x + idx_y * win_y_size] = ft_calc_color(it, color,
						it_max);
	}
}
