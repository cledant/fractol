#include "hip/hip_runtime.h"
#include "fract_ol.h"

__device__ static float	ft_mb_it(struct hipPitchedPtr *ft_matrix, size_t idx_x,
							size_t idx_y, size_t *it_max)
{
	float	it;
	float	sqrt_x;
	float	sqrt_y;
	float	tmp_x;
	float	tmp_y;

	it = 0;
	while (1)
	{
		sqrt_x = ft_matrix[idx_x][idx_y][0] * ft_matrix[idx_x][idx_y][0];
		sqrt_y = ft_matrix[idx_x][idx_y][1] * ft_matrix[idx_x][idx_y][1];
		tmp_x= sqrt_x - sqrt_y + ft_matrix[idx_x][idx_y][2];
		tmp_y = ft_matrix[idx_x][idx_y][0] * ft_matrix[idx_x][idx_y][1];
		tmp_y += tmp_y;
		tmp_y += ft_matrix[idx_x][idx_y][3];
		ft_matrix[idx_x][idx_y][0] = tmp_x; 
		ft_matrix[idx_x][idx_y][1] = tmp_y;
		if ((tmp_x * tmp_x) + (tmp_y * tmp_y) <= 4)
			return (it);
		else
			it++;
		if (it > *it_max)
			return (it);
	}
}

__device__ static unsigned int	ft_calc_color(struct hipPitchedPtr *uint_matrix,
									size_t	idx_x, size_t idx_y, int *color,
									size_t *it_max)
{
	if (uint_matrix[idx_x][idx_y][0] <= *it_max)
	{
		if (*color == 1)
			return (uint_matrix[idx_x][idx_y][0] * 0x000FF000 + 0x000000FF);
		else if (*color == 2)
			return (uint_matrix[idx_x][idx_y][0] * 0x000F0F0F + 0x00102030);
		else if (*color == 3)
			return (uint_matrix[idx_x][idx_y][0] * 0x00F0000F + 0x00111111);
		else if (*color == 4)
			return (uint_matrix[idx_x][idx_y][0] * 0x00000000 + 0x00060006);
		else if (*color == 5)
			return (uint_matrix[idx_x][idx_y][0] * 0x00A4B580 + 0x00341165);
		else if (*color == 6)
			return (uint_matrix[idx_x][idx_y][0] * 0x00000000 + 0x00060100);
	}
	else
		return (0x00000000);
	return (0x00000000);
}

__global__ void			ft_matrix_calc_mb(struct hipPitchedPtr *ft_matrix,
							struct hipPitchedPtr *uint_matrix, float *x_min,
							float *y_max, float *x_pitch, float *y_pitch,
							size_t *win_x_size, size_t *win_y_size, int *color,
							size_t *it_max)
{
	size_t	idx_x;
	size_t	idx_y;

	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x < *win_x_size && idx_y < *win_y_size)
	{
		ft_matrix[idx_x][idx_y][0] = *x_min + (idx_x * *x_pitch);
		ft_matrix[idx_x][idx_y][1] = *y_max + (idx_y * *y_pitch);
		ft_matrix[idx_x][idx_y][2] = ft_matrix[idx_x][idx_y][0];
		ft_matrix[idx_x][idx_y][3] = ft_matrix[idx_x][idx_y][1];
		ft_matrix[idx_x][idx_y][4] = ft_mb_it(ft_matrix, idx_x, idx_y);
		uint_matrix[idx_x][idx_y][0] = ft_matrix[idx_x][idx_y][4];
		uint_matrix[idx_x][idx_y][1] = ft_calc_color(uint_matrix, idx_x, idx_y,
				color, it_max);
	}
}
