#include "hip/hip_runtime.h"
extern "C"
{
#include "fract_ol.h"
}

__device__ static double		ft_abs_double(double val)
{
	return ((val < 0) ? -val : val);
}

__device__ static size_t		ft_bs_julia_it(double *pos_real, double init[2],
									size_t *it_max)
{
	size_t	it;
	double	sqrt[2];
	double	tmp[2];

	it = 0;
	while (1)
	{
		init[0] = ft_abs_double(init[0]);
		init[1] = ft_abs_double(init[1]);
		sqrt[0] = init[0] * init[0];
		sqrt[1] = init[1] * init[1];
		tmp[0]= sqrt[0] - sqrt[1] + pos_real[0];
		tmp[1] = init[0] * init[1];
		tmp[1] += tmp[1];
		tmp[1] += pos_real[1];
		init[0] = tmp[0]; 
		init[1] = tmp[1];
		if ((init[0] * init[0]) + (init[1] * init[1]) <= 4)
			it++;
		else
			return (it);
		if (it > *it_max)
			return (it);
	}
}

__global__ void			ft_matrix_calc_bs_julia(unsigned int *color_buff,
							double *x_min, double *y_max, double *x_pitch,
							double *y_pitch, size_t *win_x_size,
							size_t *win_y_size, size_t *color, size_t *it_max,
							double *mouse)
{
	size_t	idx_x;
	size_t	idx_y;
	double	init[2];
	size_t	it;

	idx_x = blockIdx.x * blockDim.x + threadIdx.x;
	idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	if (idx_x < *win_x_size && idx_y < *win_y_size)
	{
		init[0] = *x_min + (idx_x * *x_pitch);
		init[1] = -(*y_max - (idx_y * *y_pitch));
		it = ft_bs_julia_it(mouse, init, it_max);
		color_buff[idx_x + idx_y * *win_x_size] = ft_calc_color(it, color,
						it_max);
	}
}
